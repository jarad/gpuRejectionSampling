#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "cutil_inline.h"

#define THREADS_PER_BLOCK 256

__global__ void setup_prng(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void runif_kernel(hiprandState *state, double upperBound, int ni, int nd, 
                             double *uniforms, int *int_ops, double *dou_ops)
{
    int i, a, id = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    double b, u;

    // Copy state to local memory for efficiency */
    hiprandState localState = state[id];

    // Find random uniform below the upper bound 
    while ( (u=hiprand_uniform(&localState))>upperBound ) 
    {
        a=0;
        b=1;
        for (i=0; i<ni; i++) a += 1; 
        for (i=0; i<nd; i++) b *= 1.00001;
    }

    // Copy state back to global memory */
    state[id] = localState ;

    // Store results */
    uniforms[id] = u;
    int_ops[id] = a;
    dou_ops[id] = b;
}



//HIPRAND_RNG_PSEUDO_MTGP32
//__global__ void runif_kernel(int n, double ub,

extern "C" {

void gpu_runif(int *n, double *ub, int *ni, int *nd, double *u, int *nIO, double *nDO) 
{
    int nBlocks = *n/THREADS_PER_BLOCK, *d_io;
    size_t u_size = *n *sizeof(double), o_size = *n *sizeof(int);
    double *d_u, *d_do;

    cutilSafeCall( hipMalloc((void**)&d_u,  u_size) );
    cutilSafeCall( hipMalloc((void**)&d_io, o_size) );
    cutilSafeCall( hipMalloc((void**)&d_do, u_size) );

    // Setup prng states
    hiprandState *d_states;
    cutilSafeCall( hipMalloc((void**)&d_states, nBlocks*THREADS_PER_BLOCK*sizeof(hiprandState)) );
    setup_prng<<<nBlocks,THREADS_PER_BLOCK>>>(d_states);

    runif_kernel<<<nBlocks,THREADS_PER_BLOCK>>>(d_states, *ub, *ni, *nd, d_u, d_io, d_do);   
 
    cutilSafeCall( hipMemcpy(u,   d_u,  u_size, hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(nIO, d_io, o_size, hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(nDO, d_do, u_size, hipMemcpyDeviceToHost) );

    cutilSafeCall( hipFree(d_u)      );
    cutilSafeCall( hipFree(d_io)     );
    cutilSafeCall( hipFree(d_do)     );
    cutilSafeCall( hipFree(d_states) );
}

} // end of extern "C"
